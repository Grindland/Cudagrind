#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(float *a) {
   a[0] = a[0]*a[0];
}

int main() {
   float *data[2];
   float *deviceData[2];
   hipStream_t stream[2];

   srand(time(NULL));

   // Allocate memory on host and device, set host data and initialize streams
   for ( int i = 0 ; i < 2 ; i++ ) {
      hipHostMalloc(&(data[i]), sizeof(float));
      hipMalloc(&(deviceData[i]), sizeof(float));
      *(data[i]) = (rand() % 10) + (float)i+3;
      hipStreamCreate(&stream[i]);
   }

   // Print initial data
   for ( int i = 0 ; i < 2 ; i++ ) {
      printf("%f ", *(data[i]));
   }
   printf("\n");

   // Asynchronous copy of data, kernel execution and copying back of data
   for ( int i = 0 ; i < 2 ; i++ ) {
      hipMemcpyAsync(deviceData[0], data[i], sizeof(float), hipMemcpyHostToDevice, stream[i]);
      kernel<<<1, 1, 0, stream[i]>>>(deviceData[i]);
      hipMemcpyAsync(data[i], deviceData[i], sizeof(float), hipMemcpyDeviceToHost, stream[i]);
   }

   // Sync streams and print results
   for ( int i = 0 ; i < 2 ; i++ ) {
      hipStreamSynchronize(stream[i]);
      printf("%f ", *(data[i]));
   }
   printf("\n");
   
   return 0;
}



